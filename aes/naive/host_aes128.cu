#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "kernel_aes128.h"
#include "host_aes128.h"

// Key scheduling
void
h_generateCipherKey128( uchar4 *result,
						uint64_t block1,
						uint64_t block2 )
{
	for (int i = 0; i < 2; ++i) {
		result[i + 0].x = (block1 >> (56 - 32*i)) & 0xFF;
		result[i + 0].y = (block1 >> (48 - 32*i)) & 0xFF;
		result[i + 0].z = (block1 >> (40 - 32*i)) & 0xFF;
		result[i + 0].w = (block1 >> (32 - 32*i)) & 0xFF;
		
		result[i + 2].x = (block2 >> (56 - 32*i)) & 0xFF;
		result[i + 2].y = (block2 >> (48 - 32*i)) & 0xFF;
		result[i + 2].z = (block2 >> (40 - 32*i)) & 0xFF;
		result[i + 2].w = (block2 >> (32 - 32*i)) & 0xFF;
	}
}

uchar4 *
d_generateCipherKey128( uint64_t block1, uint64_t block2 )
{
	uchar4 *d_cipher_key;
	uchar4 *h_cipher_key = (uchar4 *) malloc(4 * sizeof(uchar4));
	
	// Generates the cipher key on host from the two uint64_t blocks
	h_generateCipherKey128(h_cipher_key, block1, block2);
	
	// Allocates memory for the device's cipher key, copying the host's to it
	hipMalloc((void **) &d_cipher_key,    4 * sizeof(uchar4));
	hipMemcpy(d_cipher_key, h_cipher_key, 4 * sizeof(uchar4), hipMemcpyHostToDevice);
	
	// Frees up memory used temporarily for the host cipher key
	free(h_cipher_key);
	
	return d_cipher_key;
}

uchar4 *
d_expandKey128( uchar4 *d_cipher_key, uint8_t *d_sbox ) {
	uchar4 *d_round_keys;
	
	// Allocates memory on the device for the round keys
	hipMalloc((void **) &d_round_keys, 11 * 4 * sizeof(uchar4));
	
	// Generates the round keys, storing them on the global memory
	generateRoundKeys128 <<< 1, 1 >>>
		(
			d_cipher_key, d_round_keys, d_sbox
		);
	hipDeviceSynchronize();
	
	return d_round_keys;
}


// Encryption
inline void
encryptDeviceToDevice128( char *d_contents,     uint8_t *d_sbox,
						  uchar4 *d_round_keys, size_t contents_size )
{
	// Encrypts the contents on the device
	encrypt128 <<< ((contents_size/16 + 255)/ 256), 256 >>>
		(
			d_contents, contents_size, d_round_keys, d_sbox
		);
	hipDeviceSynchronize();
}

char *
encryptHostToDevice128( char *h_contents, size_t contents_size,
						uint8_t *d_sbox, uchar4 *d_round_keys )
{
	char *d_result;
	
	// Allocates memory for the contents
	hipMalloc((void **) &d_result, contents_size);
	
	// Copies the contents to the device
	hipMemcpy(d_result, h_contents, contents_size, hipMemcpyHostToDevice);
	
	// Encrypts the contents on the device
	encryptDeviceToDevice128(d_result, d_sbox, d_round_keys, contents_size);
	
	return d_result;
}

char *
encryptHostToHost128( char *h_contents, size_t contents_size,
					  uint8_t *d_sbox,  uchar4 *d_round_keys )
{
	char *d_contents;
	char *h_result = (char *) malloc(contents_size);
	
	// Encrypts the contents on the device
	d_contents = encryptHostToDevice128( h_contents, contents_size,
										 d_sbox,     d_round_keys );
	
	// Copies back the result from the device
	hipMemcpy(h_result, d_contents, contents_size, hipMemcpyDeviceToHost);
	
	// Frees up device memory taken by the contents
	hipFree(d_contents);
	
	return h_result;
}


// Decryption
inline void
decryptDeviceToDevice128( char *d_contents,     uint8_t *d_inv_sbox,
						  uchar4 *d_round_keys, size_t contents_size )
{
	// Decrypts the contents on the device
	decrypt128 <<< ((contents_size/16 + 255)/ 256), 256 >>>
		(
			d_contents, contents_size, d_round_keys, d_inv_sbox
		);
	hipDeviceSynchronize();
}

char *
decryptHostToDevice128( char *h_contents, size_t contents_size,
						uint8_t *d_sbox,  uchar4 *d_round_keys )
{
	char *d_result;
	
	// Allocates memory for the contents
	hipMalloc((void **) &d_result, contents_size);
	
	// Copies the contents to the device
	hipMemcpy(d_result, h_contents, contents_size, hipMemcpyHostToDevice);
	
	// Encrypts the contents on the device
	decryptDeviceToDevice128(d_result, d_sbox, d_round_keys, contents_size);
	
	return d_result;
}

char *
decryptHostToHost128( char *h_contents,    size_t contents_size,
					  uint8_t *d_inv_sbox, uchar4 *d_round_keys )
{
	char *d_contents;
	char *h_result = (char *) malloc(contents_size);
	
	// Encrypts the contents on the device
	d_contents = decryptHostToDevice128( h_contents, contents_size,
										 d_inv_sbox, d_round_keys );
	
	// Copies back the result from the device
	hipMemcpy(h_result, d_contents, contents_size, hipMemcpyDeviceToHost);
	
	// Frees up device memory taken by the contents
	hipFree(d_contents);
	
	return h_result;
}
