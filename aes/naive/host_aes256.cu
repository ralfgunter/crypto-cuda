#include "hip/hip_runtime.h"
#include "kernel_aes256.h"
#include "host_aes256.h"

// Key scheduling
void
h_generateCipherKey256( uchar4 *result,
						uint64_t block1,
						uint64_t block2,
						uint64_t block3,
						uint64_t block4 )
{
	for (int i = 0; i < 2; ++i) {
		result[i + 0].x = (block1 >> (56 - 32*i)) & 0xFF;
		result[i + 0].y = (block1 >> (48 - 32*i)) & 0xFF;
		result[i + 0].z = (block1 >> (40 - 32*i)) & 0xFF;
		result[i + 0].w = (block1 >> (32 - 32*i)) & 0xFF;
		
		result[i + 2].x = (block2 >> (56 - 32*i)) & 0xFF;
		result[i + 2].y = (block2 >> (48 - 32*i)) & 0xFF;
		result[i + 2].z = (block2 >> (40 - 32*i)) & 0xFF;
		result[i + 2].w = (block2 >> (32 - 32*i)) & 0xFF;
		
		result[i + 4].x = (block3 >> (56 - 32*i)) & 0xFF;
		result[i + 4].y = (block3 >> (48 - 32*i)) & 0xFF;
		result[i + 4].z = (block3 >> (40 - 32*i)) & 0xFF;
		result[i + 4].w = (block3 >> (32 - 32*i)) & 0xFF;
		
		result[i + 6].x = (block4 >> (56 - 32*i)) & 0xFF;
		result[i + 6].y = (block4 >> (48 - 32*i)) & 0xFF;
		result[i + 6].z = (block4 >> (40 - 32*i)) & 0xFF;
		result[i + 6].w = (block4 >> (32 - 32*i)) & 0xFF;
	}
}

uchar4 *
d_generateCipherKey256( uint64_t block1,
						uint64_t block2,
						uint64_t block3,
						uint64_t block4 )
{
	uchar4 *d_cipher_key;
	uchar4 *h_cipher_key = (uchar4 *) malloc(8 * sizeof(uchar4));
	
	// Generates the cipher key on host from the four uint64_t blocks
	h_generateCipherKey256(h_cipher_key, block1, block2, block3, block4);
	
	// Allocates memory for the device's cipher key, copying the host's to it
	hipMalloc((void **) &d_cipher_key,    8 * sizeof(uchar4));
	hipMemcpy(d_cipher_key, h_cipher_key, 8 * sizeof(uchar4), hipMemcpyHostToDevice);
	
	// Frees up memory used temporarily for the host cipher key
	free(h_cipher_key);
	
	return d_cipher_key;
}

uchar4 *
d_expandKey256( uchar4 *d_cipher_key, uint8_t *d_sbox ) {
	uchar4 *d_round_keys;
	
	// Allocates memory on the device for the round keys
	hipMalloc((void **) &d_round_keys, 15 * 8 * sizeof(uchar4));
	
	// Generates the round keys, storing them on the global memory
	generateRoundKeys256 <<< 1, 1 >>>
		(
			d_cipher_key, d_round_keys, d_sbox
		);
	hipDeviceSynchronize();
	
	return d_round_keys;
}


// Encryption
inline void
encryptDeviceToDevice256( char *d_contents,     uint8_t *d_sbox,
						  uchar4 *d_round_keys, size_t contents_size )
{
	// Encrypts the contents on the device
	encrypt256 <<< ((contents_size/16 + 255)/ 256), 256 >>>
		(
			d_contents, contents_size, d_round_keys, d_sbox
		);
	hipDeviceSynchronize();
}

char *
encryptHostToDevice256( char *h_contents, size_t contents_size,
						uint8_t *d_sbox, uchar4 *d_round_keys )
{
	char *d_result;
	
	// Allocates memory for the contents
	hipMalloc((void **) &d_result, contents_size);
	
	// Copies the contents to the device
	hipMemcpy(d_result, h_contents, contents_size, hipMemcpyHostToDevice);
	
	// Encrypts the contents on the device
	encryptDeviceToDevice256(d_result, d_sbox, d_round_keys, contents_size);
	
	return d_result;
}

char *
encryptHostToHost256( char *h_contents, size_t contents_size,
					  uint8_t *d_sbox,  uchar4 *d_round_keys )
{
	char *d_contents;
	char *h_result = (char *) malloc(contents_size);
	
	// Encrypts the contents on the device
	d_contents = encryptHostToDevice256( h_contents, contents_size,
										 d_sbox,     d_round_keys );
	
	// Copies back the result from the device
	hipMemcpy(h_result, d_contents, contents_size, hipMemcpyDeviceToHost);
	
	// Frees up device memory taken by the contents
	hipFree(d_contents);
	
	return h_result;
}


// Decryption
inline void
decryptDeviceToDevice256( char *d_contents,     uint8_t *d_inv_sbox,
						  uchar4 *d_round_keys, size_t contents_size )
{
	// Decrypts the contents on the device
	decrypt256 <<< ((contents_size/16 + 255)/ 256), 256 >>>
		(
			d_contents, contents_size, d_round_keys, d_inv_sbox
		);
	hipDeviceSynchronize();
}

char *
decryptHostToDevice256( char *h_contents, size_t contents_size,
						uint8_t *d_sbox,  uchar4 *d_round_keys )
{
	char *d_result;
	
	// Allocates memory for the contents
	hipMalloc((void **) &d_result, contents_size);
	
	// Copies the contents to the device
	hipMemcpy(d_result, h_contents, contents_size, hipMemcpyHostToDevice);
	
	// Encrypts the contents on the device
	decryptDeviceToDevice256(d_result, d_sbox, d_round_keys, contents_size);
	
	return d_result;
}

char *
decryptHostToHost256( char *h_contents,     size_t contents_size,
					  uint8_t *d_inv_sbox,  uchar4 *d_round_keys )
{
	char *d_contents;
	char *h_result = (char *) malloc(contents_size);
	
	// Encrypts the contents on the device
	d_contents = decryptHostToDevice256( h_contents, contents_size,
										 d_inv_sbox, d_round_keys );
	
	// Copies back the result from the device
	hipMemcpy(h_result, d_contents, contents_size, hipMemcpyDeviceToHost);
	
	// Frees up device memory taken by the contents
	hipFree(d_contents);
	
	return h_result;
}
